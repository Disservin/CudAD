#include "hip/hip_runtime.h"
#include "activations/ClippedReLU.h"
#include "activations/Linear.h"
#include "activations/ReLU.h"
#include "activations/Sigmoid.h"
#include "data/DenseMatrix.h"
#include "data/SArray.h"
#include "data/Tape.h"
#include "dataset/batchloader.h"
#include "dataset/dataset.h"
#include "dataset/io.h"
#include "dataset/reader.h"
#include "dataset/writer.h"
#include "dataset/shuffle.h"
#include "layer/DenseLayer.h"
#include "layer/DuplicateDenseLayer.h"
#include "loss/MLE.h"
#include "loss/MPE.h"
#include "loss/MSE.h"
#include "mappings.h"
#include "misc/timer.h"
#include "network/Network.h"
#include "operations/operations.h"
#include "optimizer/Adam.h"
#include "position/fenparsing.h"
#include "position/position.h"
#include "quantitize.h"
#include "misc/csv.h"

#include <filesystem>
#include <iostream>

const std::string data_path = "H:/Koivisto Resourcen/Training Data 7.9/";
std::string output = "../resources/runs/experiment_22/";

float validate(Network&     network,
                           DataSet&     data_set,
                           DenseMatrix& target,
                           SArray<bool>& target_mask,
                           SparseInput& i1,
                           SparseInput& i2) {

    int B = i1.n;

    // reset loss
    float prev_loss = network.getLossFunction()->getLoss().get(0);
    network.getLossFunction()->getLoss().get(0) = 0;
    network.getLossFunction()->getLoss().gpu_upload();

    int c = std::floor(data_set.positions.size() / B);
    for(int i = 0; i < c; i++){
        int id1 = i   * B;
        int id2 = id1 + B;
        DataSet temp{};
        temp.header.position_count = B;
        temp.positions.assign(&data_set.positions[id1],&data_set.positions[id2]);

        dense_berky::assign_inputs_batch(temp, i1, i2, target, target_mask);

        i1.column_indices.gpu_upload();
        i2.column_indices.gpu_upload();
        target.gpu_upload();
        target_mask.gpu_upload();

        network.feed(std::vector<SparseInput*> {&i1, &i2});

        network.getLossFunction()->apply(network.getOutput().values,
                                         network.getOutput().gradients,
                                         target,
                                         target_mask,
                                         DEVICE);
    }

    network.getLossFunction()->getLoss().gpu_download();
    float loss = network.getLossFunction()->getLoss().get(0);

    network.getLossFunction()->getLoss().get(0) = prev_loss;
    network.getLossFunction()->getLoss().gpu_upload();

    return loss / c;
}
int main() {
    init();

    // definitions
    constexpr uint32_t       I = 16 * 12 * 64;
    constexpr uint32_t       H = 512;
    constexpr uint32_t       H2= 16;
    constexpr uint32_t       O = 1;
    constexpr uint32_t       B = 16384;
    constexpr uint32_t     BPE = 6100;
    constexpr  int32_t       E = 600;

    // Load files
    std::vector<std::string> files {};
    for (int i = 0; i < 24; i++)
        files.push_back(data_path + "reshuffled_generated_" + std::to_string(i) + ".txt.bin");

    BatchLoader  batch_loader {files, B};

    // Input data (perspective)
    SparseInput  i0 {I, B, 32};    // 32 max inputs
    SparseInput  i1 {I, B, 32};

    DenseMatrix  target {O, B};
    SArray<bool> target_mask {O * B};
    target_mask.malloc_cpu();
    target_mask.malloc_gpu();

    // 1536 -> (2x512) -> 1
    DuplicateDenseLayer<I, H, Linear> l1 {};
    DenseLayer<H * 2, H2, ReLU>   l2 {};
    DenseLayer<H2, O, Linear>   l3 {};
//    l2.getTunableParameters()[0]->min_allowed_value = 0;
//    l2.getTunableParameters()[0]->max_allowed_value = 1;

    // Berserk sigmoid
//    dynamic_cast<Sigmoid*>(l3.getActivationFunction())->scalar = 2.5 / 400;

//    l1.lasso_regularization = (1.0 / 8388608.0);

    // stack layers to build network
    std::vector<LayerInterface*> layers {};
    layers.push_back(&l1);
    layers.push_back(&l2);
    layers.push_back(&l3);

    Network network {layers};

    // loss function
    MPE     loss_function {2.5, true};
    network.setLossFunction(&loss_function);

    // optimizer
    Adam adam {};
    adam.init(layers);
    adam.alpha = 0.0001;
    adam.beta1 = 0.95;
    adam.beta2 = 0.999;

    network.loadWeights(output + "networks/weights-epoch" + std::to_string(600) + ".nn");
    computeScalars(batch_loader, network, 8192, I);
//    test_fen(network, "rnbqkbnr/pppppppp/8/8/8/8/PPPPPPPP/RNBQKBNR w QKqk - 0 1", I);
//    quantitize(output + std::to_string(600) + ".net", network, 16, 256);

//    Timer t {};
//    CSVWriter csv_writer{output + "loss.csv"};
////    logging::open(output + "loss.csv");
//    for (int epoch = 1; epoch <= E; epoch++) {
//        float epoch_loss = 0;
//        t.tick();
//        for (int batch = 0; batch < BPE; batch++) {
//
//            // get the next dataset (batch)
//            auto* ds = batch_loader.next();
//            // assign to the inputs and compute the target
//            dense_relative::assign_inputs_batch(*ds, i0, i1, target, target_mask);
//            // upload relevant data
//            i0.column_indices.gpu_upload();
//            i1.column_indices.gpu_upload();
//            target.gpu_upload();
//            target_mask.gpu_upload();
//
//            // download the loss to display the loss of the iteration
//            loss_function.loss.gpu_download();
//
//            // measure time and print output
//            t.tock();
//            std::printf("\rep/ba = [%3d/%5d], ", epoch, batch + 1);
//            std::printf("batch_loss = [%1.8f], ", loss_function.loss(0));
//            std::printf("epoch_loss = [%1.8f], ", epoch_loss / (batch + 1));
//            std::printf("speed = [%9d pos/s], ", (int) std::round(1000.0f * B * (batch + 1) / t.duration()));
//            std::printf("time = [%3ds]", (int) t.duration() / 1000);
//            std::cout << std::flush;
//
//            epoch_loss += loss_function.loss(0);
//            // make sure to reset the loss here since the mse increments the loss in order to not have
//            // to use memcpy (might change soon)
//            loss_function.loss(0) = 0;
//            loss_function.loss.gpu_upload();
//
//            // feed forward
//            network.batch(std::vector<SparseInput*> {&i0, &i1}, target, target_mask);
//
//            // update weights
//            adam.apply(1);
//        }
//        std::cout << std::endl;
//        csv_writer.write({std::to_string(epoch),
//                          std::to_string(epoch_loss / BPE)});
//
//        network.saveWeights(output + "networks/weights-epoch" + std::to_string(epoch) + ".nn");
//
//        computeScalars(batch_loader, network, 128, I);
//        if (epoch % (29 * 10) == 0)
//            adam.alpha *= 0.1f;
//        if (epoch == 3){
//            adam.alpha = 0.01;
//        }
//    }

    close();
}
