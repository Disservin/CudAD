#include "hip/hip_runtime.h"
#include "activations/ClippedReLU.h"
#include "activations/Linear.h"
#include "activations/ReLU.h"
#include "activations/Sigmoid.h"
#include "data/DenseMatrix.h"
#include "data/SArray.h"
#include "data/Tape.h"
#include "dataset/batchloader.h"
#include "dataset/dataset.h"
#include "dataset/io.h"
#include "dataset/reader.h"
#include "dataset/writer.h"
#include "layer/DenseLayer.h"
#include "layer/DuplicateDenseLayer.h"
#include "loss/MLE.h"
#include "loss/MPE.h"
#include "loss/MSE.h"
#include "mappings.h"
#include "misc/timer.h"
#include "network/Network.h"
#include "operations/operations.h"
#include "optimizer/Adam.h"
#include "position/fenparsing.h"
#include "position/position.h"
#include "quantitize.h"

#include <filesystem>
#include <iostream>

const std::string data_path = "E:/berserk/training-data/berserk9dev2/finny-data/";
const std::string output = "./resources/runs/experiment_3/";

int main() {

    init();

    // definitions
    constexpr uint32_t       I = 12 * 2 * 64;
    constexpr uint32_t       H = 512;
    constexpr uint32_t       O = 1;
    constexpr uint32_t       B = 16384;
    constexpr uint32_t BATCHES_PER_EPOCH = 6100;


    // Load files
    std::vector<std::string> files {};
    for (int i = 0; i < 7; i++)
        files.push_back(data_path + "berserk9dev2.d9." + std::to_string(i) + ".bin");

    BatchLoader  batch_loader {files, B, 1};

    // Input data (perspective)
    SparseInput  sparse_input_1 {I, B, 32};    // 32 max inputs
    SparseInput  sparse_input_2 {I, B, 32};

    DenseMatrix  target {O, B};
    SArray<bool> target_mask {O * B};
    target_mask.malloc_cpu();
    target_mask.malloc_gpu();

    // 1536 -> (2x512) -> 1
    DuplicateDenseLayer<I, H, ReLU> l1 {};
    DenseLayer<H * 2, O, Sigmoid>   l2 {};

    // Berserk sigmoid
    dynamic_cast<Sigmoid*>(l2.getActivationFunction())->scalar = 3.68415f / 512;

    // stack layers to build network
    std::vector<LayerInterface*> layers {};
    layers.push_back(&l1);
    layers.push_back(&l2);

    Network network {layers};

    // loss function
    MPE     loss_function {2.5};
    network.setLossFunction(&loss_function);

    // optimizer
    Adam adam {};
    adam.init(layers);
    adam.alpha = 0.01;
    adam.beta1 = 0.9f;
    adam.beta2 = 0.999f;

    logging::open(output + "loss.csv");

    Timer t {};
    for (int epoch = 1; epoch <= 420; epoch++) {
        float epoch_loss = 0;
        t.tick();
        for (int batch = 0; batch < BATCHES_PER_EPOCH; batch++) {

            // get the next dataset (batch)
            auto* ds = batch_loader.next();
            // assign to the inputs and compute the target
            dense_berky::assign_inputs_batch(*ds,
                                                sparse_input_1,
                                                sparse_input_2,
                                                target,
                                                target_mask);
            // upload relevant data
            sparse_input_1.column_indices.gpu_upload();
            sparse_input_2.column_indices.gpu_upload();
            target.gpu_upload();
            target_mask.gpu_upload();

            // download the loss to display the loss of the iteration
            loss_function.loss.gpu_download();

            // measure time
            t.tock();
            std::stringstream ss {};
            ss << "    ep/ba = [" << std::setw(4) << epoch << "/" << std::setw(5) << batch << "]"
               << "    ba. loss  = " << std::setw(14) << loss_function.loss(0)
               << "    ep. loss  = " << std::setw(14) << epoch_loss / (batch + 1)
               << "    speed = " << std::setw(10)
               << std::round(1000.0f * (B * (batch + 1) / (float) t.duration()));
            std::cout << "\r" << ss.str() << std::flush;

            epoch_loss += loss_function.loss(0);
            // make sure to reset the loss here since the mse increments the loss in order to not have
            // to use memcpy (might change soon)
            loss_function.loss(0) = 0;
            loss_function.loss.gpu_upload();

            // feed forward
            network.batch(std::vector<SparseInput*> {&sparse_input_1, &sparse_input_2},
                          target,
                          target_mask);

            // update weights
            adam.apply(1);
        }
        std::cout << std::endl;
        logging::write("\"" + std::to_string(epoch) + "\",\"" + std::to_string(epoch_loss / BATCHES_PER_EPOCH) + "\"");

        network.saveWeights(output + "weights-epoch" + std::to_string(epoch) + ".nn");
        quantitize(output + "nn-epoch" + std::to_string(epoch) + ".nnue", network);

        if (epoch % 105 == 0)
            adam.alpha *= 0.1f;
    }

    close();
}
