#include "hip/hip_runtime.h"

//
// Created by Luecx on 14.01.2022.
//

#include "sparse_affine_bp.h"
__global__ void sparse_affine_bp_kernel(
          float*        __restrict__ mat_grd,
    const unsigned int* __restrict__ inp_col_indices,
    const unsigned int               inp_col_max_entries,
          float*        __restrict__ bia_grd,
    const float*        __restrict__ res_grd,
    const unsigned int               m,
    const unsigned int               n,
    const unsigned int               lda,
    const unsigned int               ldc){

    // compute which output value we are looking at
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // skip out of bounds
    if (col >= n || row >= m) return;

    // get the offset at which we look into our sparse input
    int offset = col * (inp_col_max_entries + 1);
    // check how many values we are going to read
    int count = inp_col_indices[offset];

    // track the sum
    float res_grd_v = res_grd[MATRIX_INDEX(ldc, row, col)];

    atomicAdd(&bia_grd[row], res_grd_v);

    // start at offset + 1 (offset contains the amount of values to read)
    for (int i = offset + 1; i < offset + 1 + count; i++) {

        // get the sparse index (set row of the input)
        auto b_row = inp_col_indices[i];
        // get the corresponding weight
        atomicAdd(&mat_grd[MATRIX_INDEX(lda, row, b_row)], res_grd_v);
    }
};