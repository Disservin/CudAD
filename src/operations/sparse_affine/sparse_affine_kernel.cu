#include "hip/hip_runtime.h"
/**
    CudAD is a CUDA neural network trainer, specific for chess engines.
    Copyright (C) 2022 Finn Eggers

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "sparse_affine.h"
// clang-format off
__global__ void sparse_affine_kernel(
    const float*        __restrict__ mat,
    const unsigned int* __restrict__ inp_col_indices,
    const unsigned int               inp_col_max_entries,
    const float*        __restrict__ bia,
          float*        __restrict__ res,
    const unsigned int               m,
    const unsigned int               n,
    const unsigned int               lda,
    const unsigned int               ldc){

    // clang-format on
    // compute which output value we are looking at
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // skip out of bounds
    if (col >= n || row >= m)
        return;

    // get the offset at which we look into our sparse input
    int offset = col * (inp_col_max_entries + 1);
    // check how many values we are going to read
    int count = inp_col_indices[offset];

    // track the sum
    float sum = bia[row];

    // start at offset + 1 (offset contains the amount of values to read)
    for (int i = offset + 1; i < offset + 1 + count; i++) {

        // get the sparse index (set row of the input)
        auto b_row = inp_col_indices[i];
        // get the corresponding weight
        auto wgt = mat[MATRIX_INDEX(lda, row, b_row)];
        sum += wgt;
    }
    res[MATRIX_INDEX(ldc, row, col)] = sum;
};