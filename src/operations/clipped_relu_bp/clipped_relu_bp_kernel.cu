#include "hip/hip_runtime.h"

//
// Created by Luecx on 13.01.2022.
//
#include "clipped_relu_bp.h"
/**
 * performs C = alpha * A + beta * B
 * @param A
 * @param B
 * @param C
 * @param size
 * @param alpha
 * @param beta
 * @return
 */
__global__ void clipped_relu_bp_kernel(
    const float* __restrict__ A,
          float* __restrict__ A_grd,
    const float* __restrict__ B,
    const float* __restrict__ B_grd,
    unsigned int size,
    float mmax){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= size) return;

    if(A[idx] > 0 && A[idx] < mmax){
        A_grd[idx] = B_grd[idx];
    }else{
        A_grd[idx] = 0;
    }
}
