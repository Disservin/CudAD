
//
// Created by Luecx on 13.01.2022.
//

#include <hip/hip_runtime.h>
#include <iostream>
/**
 * performs C = alpha * A + beta * B
 * @param A
 * @param B
 * @param C
 * @param size
 * @param alpha
 * @param beta
 */
void clipped_relu_bp_host(
    const float* A,
          float* A_grd,
    const float* B,
    const float* B_grd,
    unsigned int size,
    float max){

    for(int idx = 0; idx < size; idx++){
        if(A[idx] > 0 && A[idx] < max){
            A_grd[idx] = B_grd[idx];
        }else{
            A_grd[idx] = 0;
        }
    }
}
