#include "hip/hip_runtime.h"

//
// Created by Luecx on 13.01.2022.
//
#include "clipped_relu_bp.h"

#include <iostream>

// clang-format off
void clipped_relu_bp_host(
    const float* A,
          float* A_grd,
    const float* B,
    const float* B_grd,
    unsigned int size,
    float max){
    // clang-format on

    for (int idx = 0; idx < size; idx++) {
        if (B[idx] > 0 && B[idx] < max) {
            A_grd[idx] = B_grd[idx];
        } else {
            A_grd[idx] = 0;
        }
    }
}
