
//
// Created by Luecx on 13.01.2022.
//

#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
/**
 * performs C = alpha * A + beta * B
 * @param A
 * @param B
 * @param C
 * @param size
 * @param alpha
 * @param beta
 */
void clipped_relu_host(
    const float* A,
          float* B,
    unsigned int size,
    float max){

    for(int i = 0; i < size; i++){
        B[i] = std::min(max, std::max(A[i], 0.0f));
    }
}
