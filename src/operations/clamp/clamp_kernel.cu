
#include <hip/hip_runtime.h>

//
// Created by Luecx on 13.01.2022.
//

__global__ void clamp_kernel(
    float* __restrict__ values,
    float p_min,
    float p_max,
    float size){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= size) return;

    values[idx] = max(p_min, min(p_max, values[idx]));
}