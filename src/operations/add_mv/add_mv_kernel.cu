#include "hip/hip_runtime.h"

//
// Created by Luecx on 13.01.2022.
//
#include "add_mv.h"
/**
 * performs C = alpha * A + beta * B
 * @param A
 * @param B
 * @param C
 * @param size
 * @param alpha
 * @param beta
 * @return
 */
__global__ void add_mv_kernel(
    const float* __restrict__ mat,
    const float* __restrict__ vec,
          float* __restrict__ res,
          int m,
          int n,
          int ld_mat,
          int ld_res){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if(idx >= n || idy >= m) return;

    res[MATRIX_INDEX(ld_res, idy, idx)] = mat[MATRIX_INDEX(ld_mat, idy, idx)] + vec[idy];
}
