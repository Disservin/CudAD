
//
// Created by Luecx on 13.01.2022.
//

#include <hip/hip_runtime.h>
#include <iostream>
/**
 * performs C = alpha * A + beta * B
 * @param A
 * @param B
 * @param C
 * @param size
 * @param alpha
 * @param beta
 */
void relu_bp_host(
    const float* A,
          float* A_grd,
    const float* B,
    const float* B_grd,
    unsigned int size){

    for(int idx = 0; idx < size; idx++){
        if(B[idx] > 0){
            A_grd[idx] = B_grd[idx];
        }else{
            A_grd[idx] = 0;
        }
    }
}
