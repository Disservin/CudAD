
//
// Created by Luecx on 13.01.2022.
//

#include <hip/hip_runtime.h>
#include <iostream>
/**
 * performs C = alpha * A + beta * B
 * @param A
 * @param B
 * @param C
 * @param size
 * @param alpha
 * @param beta
 */
void sigmoid_bp_host(
    const float* A,
          float* A_grd,
    const float* B,
    const float* B_grd,
    unsigned int size,
    float scalar){

    for(int idx = 0; idx < size; idx++){
        A_grd[idx] = B_grd[idx] * B[idx] * (1 - B[idx]) * scalar;
    }
}
