#include "hip/hip_runtime.h"

//
// Created by Luecx on 13.01.2022.
//
#include "relu.h"

// clang-format off
__global__ void relu_kernel(
    const float* __restrict__ A,
          float* __restrict__ B,
    unsigned int size){
    // clang-format on

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= size)
        return;

    B[idx] = max(0.0f, A[idx]);
}
